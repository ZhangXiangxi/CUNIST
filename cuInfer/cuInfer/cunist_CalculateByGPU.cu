#include "hip/hip_runtime.h"
#include "cunist_CalculateByGPU.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>
#define BW 128

using namespace std;

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator) {
	return (nominator + denominator - 1) / denominator;
}

/**
* Computes the backpropagation results of the Softmax loss for each result in a batch.
* Uses the softmax values obtained from forward propagation to compute the difference.
*
* @param label The training batch label values.
* @param num_labels The number of possible labels.
* @param batch_size The size of the trained batch.
* @param diff The resulting gradient.
*/
__global__ void SoftmaxLossBackprop(const float* label, int num_labels, int batch_size, float* diff) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch_size)
		return;

	const int label_value = static_cast<int>(label[idx]);

	// For each item in the batch, decrease the result of the label's value by 1
	diff[idx * num_labels + label_value] -= 1.0f;
}

const int FLAGS_gpu = 0;				// The GPU ID to use
const int FLAGS_iterations = 6000;		// Number of iterations for training

// Batch parameters
const int FLAGS_batch_size = 64;		// Batch size for training


// Filenames

const string FLAGS_train_images("train-images.idx3-ubyte");	// Training images filename
const string FLAGS_train_labels("train-labels.idx1-ubyte");	// Training labels filename
const string FLAGS_test_images("t10k-images.idx3-ubyte");		// Test images filename
const string FLAGS_test_labels("t10k-labels.idx1-ubyte");		// Test labels filename
const string DATA_FILE_NAME("D:\\IntervalData.dat");					// Interval Data Filename

// Solver parameters
const double FLAGS_learning_rate = 0.01;		// Base learning rate
const double FLAGS_lr_gamma = 0.0001;			// Learning rate policy gamma
const double FLAGS_lr_power = 0.75;				// Learning rate policy power

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code 
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
		} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
				    }                                                  \
		} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
				    }                                                  \
		} while(0)

///////////////////////////////////////////////////////////////////////////////////////////
// Layer representations

/**
* Represents a convolutional layer with bias.
*/
struct ConvBiasLayer {
	int in_channels, out_channels, kernel_size;
	int in_width, in_height, out_width, out_height;

	vector<float> pconv, pbias;

	ConvBiasLayer(int in_channels_, int out_channels_, int kernel_size_,
		int in_w_, int in_h_) : pconv(in_channels_ * kernel_size_ * kernel_size_ * out_channels_),
		pbias(out_channels_) {
		in_channels = in_channels_;
		out_channels = out_channels_;
		kernel_size = kernel_size_;
		in_width = in_w_;
		in_height = in_h_;
		out_width = in_w_ - kernel_size_ + 1;
		out_height = in_h_ - kernel_size_ + 1;
	}
};

/**
* Represents a max-pooling layer.
*/
struct MaxPoolLayer {
	int size, stride;

	MaxPoolLayer(int size_, int stride_) : size(size_), stride(stride_) {
	}
};

/**
* Represents a fully-connected neural network layer with bias.
*/
struct FullyConnectedLayer {
	int inputs, outputs;
	std::vector<float> pneurons, pbias;

	FullyConnectedLayer(int inputs_, int outputs_) : inputs(inputs_), outputs(outputs_),
		pneurons(inputs_ * outputs_), pbias(outputs_) {
	}
};

struct TrainingContext {
	hipdnnHandle_t cudnnHandle;
	hipblasHandle_t cublasHandle;

	hipdnnTensorDescriptor_t dataTensor, conv1Tensor, conv1BiasTensor, pool1Tensor,
		conv2Tensor, conv2BiasTensor, pool2Tensor, fc1Tensor, fc2Tensor;
	hipdnnFilterDescriptor_t conv1filterDesc, conv2filterDesc;
	hipdnnConvolutionDescriptor_t conv1Desc, conv2Desc;
	hipdnnConvolutionFwdAlgo_t conv1algo, conv2algo;
	hipdnnPoolingDescriptor_t poolDesc;

	int m_gpuid;
	int m_batchSize;
	size_t m_workspaceSize;

	FullyConnectedLayer &ref_fc1, &ref_fc2;

	// Disable copying
	TrainingContext& operator=(const TrainingContext&) = delete;
	TrainingContext(const TrainingContext&) = delete;

	TrainingContext(int gpuid, int batch_size,
		ConvBiasLayer& conv1, MaxPoolLayer& pool1, ConvBiasLayer& conv2, MaxPoolLayer& pool2,
		FullyConnectedLayer& fc1, FullyConnectedLayer& fc2) : m_gpuid(gpuid), ref_fc1(fc1), ref_fc2(fc2) {
		m_batchSize = batch_size;

		// Create CUBLAS and CUDNN handles
		checkCudaErrors(hipSetDevice(gpuid));
		checkCudaErrors(hipblasCreate(&cublasHandle));
		checkCUDNN(hipdnnCreate(&cudnnHandle));

		// Create tensor descriptors
		checkCUDNN(hipdnnCreateTensorDescriptor(&dataTensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv1Tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv1BiasTensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&pool1Tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv2Tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&conv2BiasTensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&pool2Tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&fc1Tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&fc2Tensor));

		checkCUDNN(hipdnnCreateFilterDescriptor(&conv1filterDesc));
		checkCUDNN(hipdnnCreateFilterDescriptor(&conv2filterDesc));

		checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv1Desc));
		checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv2Desc));

		checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));


		// Set tensor descriptor sizes
		checkCUDNN(hipdnnSetTensor4dDescriptor(conv1BiasTensor,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			1, conv1.out_channels,
			1, 1));
		checkCUDNN(hipdnnSetTensor4dDescriptor(conv2BiasTensor,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			1, conv2.out_channels,
			1, 1));

		checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc,
			HIPDNN_POOLING_MAX,
			pool1.size, pool1.size,
			0, 0,
			pool1.stride, pool1.stride));
		checkCUDNN(hipdnnSetTensor4dDescriptor(pool2Tensor,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batch_size, conv2.out_channels,
			conv2.out_height / pool2.stride,
			conv2.out_width / pool2.stride));

		checkCUDNN(hipdnnSetTensor4dDescriptor(fc1Tensor,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batch_size, fc1.outputs, 1, 1));

		checkCUDNN(hipdnnSetTensor4dDescriptor(fc2Tensor,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batch_size, fc2.outputs, 1, 1));


		// Set convolution tensor sizes and compute workspace size
		size_t workspace = 0;
		workspace = std::max(workspace, SetFwdConvolutionTensors(conv1, dataTensor, conv1Tensor, conv1filterDesc, conv1Desc, conv1algo));
		workspace = std::max(workspace, SetFwdConvolutionTensors(conv2, pool1Tensor, conv2Tensor, conv2filterDesc, conv2Desc, conv2algo));

		// The workspace is allocated later (if necessary)
		m_workspaceSize = workspace;
	}

	~TrainingContext() {
		checkCudaErrors(hipSetDevice(m_gpuid));

		checkCudaErrors(hipblasDestroy(cublasHandle));
		checkCUDNN(hipdnnDestroy(cudnnHandle));
		checkCUDNN(hipdnnDestroyTensorDescriptor(dataTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv1Tensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv1BiasTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(pool1Tensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv2Tensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(conv2BiasTensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(pool2Tensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(fc1Tensor));
		checkCUDNN(hipdnnDestroyTensorDescriptor(fc2Tensor));
		checkCUDNN(hipdnnDestroyFilterDescriptor(conv1filterDesc));
		checkCUDNN(hipdnnDestroyFilterDescriptor(conv2filterDesc));
		checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv1Desc));
		checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv2Desc));
		checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
	}

	size_t SetFwdConvolutionTensors(ConvBiasLayer& conv, hipdnnTensorDescriptor_t& srcTensorDesc, hipdnnTensorDescriptor_t& dstTensorDesc,
		hipdnnFilterDescriptor_t& filterDesc, hipdnnConvolutionDescriptor_t& convDesc,
		hipdnnConvolutionFwdAlgo_t& algo) {
		size_t sizeInBytes = 0;

		int n = m_batchSize;
		int c = conv.in_channels;
		int h = conv.in_height;
		int w = conv.in_width;

		checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			n, c,
			h, w));

		checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
			HIPDNN_DATA_FLOAT,
			conv.out_channels,
			conv.in_channels,
			conv.kernel_size,
			conv.kernel_size));

		checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
			0, 0,
			1, 1,
			1, 1,
			HIPDNN_CROSS_CORRELATION));
		// Find dimension of convolution output
		checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
			srcTensorDesc,
			filterDesc,
			&n, &c, &h, &w));

		checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			n, c,
			h, w));
		checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
			srcTensorDesc,
			filterDesc,
			convDesc,
			dstTensorDesc,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
			0,
			&algo));

		checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
			srcTensorDesc,
			filterDesc,
			convDesc,
			dstTensorDesc,
			algo,
			&sizeInBytes));

		return sizeInBytes;
	}

	void ForwardPropagation(float* data, float* conv1, float* pool1, float* conv2, float* pool2, float* fc1, float* fc1relu,
		float* fc2, float* result,
		float* pconv1, float* pconv1bias,
		float* pconv2, float* pconv2bias,
		float* pfc1, float* pfc1bias,
		float* pfc2, float* pfc2bias, void* workspace, float* onevec) {
		float alpha = 1.0f, beta = 0.0f;
		checkCudaErrors(hipSetDevice(m_gpuid));

		// Conv1 layer
		checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, dataTensor,
			data, conv1filterDesc, pconv1, conv1Desc,
			conv1algo, workspace, m_workspaceSize, &beta,
			conv1Tensor, conv1));
		checkCUDNN(hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C, &alpha, conv1BiasTensor,
			pconv1bias, &alpha, conv1Tensor, conv1));

		// Pool1 layer
		checkCUDNN(hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, conv1Tensor,
			conv1, &beta, pool1Tensor, pool1));

		// Conv2 layer
		checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, pool1Tensor,
			pool1, conv2filterDesc, pconv2, conv2Desc,
			conv2algo, workspace, m_workspaceSize, &beta,
			conv2Tensor, conv2));
		checkCUDNN(hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C, &alpha, conv2BiasTensor,
			pconv2bias, &alpha, conv2Tensor, conv2));

		// Pool2 layer
		checkCUDNN(hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, conv2Tensor,
			conv2, &beta, pool2Tensor, pool2));

		// FC1 layer
		// Forward propagate neurons using weights (fc1 = pfc1'*pool2)
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			ref_fc1.outputs, m_batchSize, ref_fc1.inputs,
			&alpha,
			pfc1, ref_fc1.inputs,
			pool2, ref_fc1.inputs,
			&beta,
			fc1, ref_fc1.outputs));
		// Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			ref_fc1.outputs, m_batchSize, 1,
			&alpha,
			pfc1bias, ref_fc1.outputs,
			onevec, 1,
			&alpha,
			fc1, ref_fc1.outputs));

		// ReLU activation
		checkCUDNN(hipdnnActivationForward(cudnnHandle, HIPDNN_ACTIVATION_RELU, &alpha,
			fc1Tensor, fc1, &beta, fc1Tensor, fc1relu));

		// FC2 layer
		// Forward propagate neurons using weights (fc2 = pfc2'*fc1relu)
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			ref_fc2.outputs, m_batchSize, ref_fc2.inputs,
			&alpha,
			pfc2, ref_fc2.inputs,
			fc1relu, ref_fc2.inputs,
			&beta,
			fc2, ref_fc2.outputs));
		// Add bias using GEMM's "beta" (fc2 += pfc2bias*1_vec')
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			ref_fc2.outputs, m_batchSize, 1,
			&alpha,
			pfc2bias, ref_fc2.outputs,
			onevec, 1,
			&alpha,
			fc2, ref_fc2.outputs));

		// Softmax loss
		checkCUDNN(hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
			&alpha, fc2Tensor, fc2, &beta, fc2Tensor, result));
	}

	void Backpropagation(ConvBiasLayer& layer_conv1, MaxPoolLayer& layer_pool1, ConvBiasLayer& layer_conv2, MaxPoolLayer& layer_pool2,
		float* data, float* labels, float* conv1, float* pool1, float* conv2, float* pool2, float* fc1, float* fc1relu,
		float* fc2, float* fc2smax, float* dloss_data,
		float* pconv1, float* pconv1bias,
		float* pconv2, float* pconv2bias,
		float* pfc1, float* pfc1bias,
		float* pfc2, float* pfc2bias,
		float* gconv1, float* gconv1bias, float* dpool1,
		float* gconv2, float* gconv2bias, float* dconv2, float* dpool2,
		float* gfc1, float* gfc1bias, float* dfc1, float* dfc1relu,
		float* gfc2, float* gfc2bias, float* dfc2,
		float* onevec) {
		float alpha = 1.0f, beta = 0.0f;

		float scalVal = 1.0f / static_cast<float>(m_batchSize);

		checkCudaErrors(hipSetDevice(m_gpuid));

		// Initialization (using the training error function)
		checkCudaErrors(hipMemcpyAsync(dloss_data, fc2smax, sizeof(float) * m_batchSize * ref_fc2.outputs, hipMemcpyDeviceToDevice));

		// Softmax layer
		SoftmaxLossBackprop << <RoundUp(m_batchSize, BW), BW >> >(labels, ref_fc2.outputs, m_batchSize, dloss_data);

		// Accounting for batch size in SGD
		checkCudaErrors(hipblasSscal(cublasHandle, ref_fc2.outputs * m_batchSize, &scalVal, dloss_data, 1));

		// FC2 layer
		// Compute derivative with respect to weights: gfc2 = (fc1relu * dfc2smax')
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_fc2.inputs, ref_fc2.outputs, m_batchSize,
			&alpha, fc1relu, ref_fc2.inputs, dloss_data, ref_fc2.outputs, &beta, gfc2, ref_fc2.inputs));
		// Compute derivative with respect to bias: gfc2bias = dfc2smax * 1_vec
		checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_fc2.outputs, m_batchSize,
			&alpha, dloss_data, ref_fc2.outputs, onevec, 1, &beta, gfc2bias, 1));
		// Compute derivative with respect to data (for previous layer): pfc2*dfc2smax (500x10*10xN)
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_fc2.inputs, m_batchSize, ref_fc2.outputs,
			&alpha, pfc2, ref_fc2.inputs, dloss_data, ref_fc2.outputs, &beta, dfc2, ref_fc2.inputs));

		// ReLU activation
		checkCUDNN(hipdnnActivationBackward(cudnnHandle, HIPDNN_ACTIVATION_RELU, &alpha,
			fc1Tensor, fc1relu, fc1Tensor, dfc2,
			fc1Tensor, fc1, &beta, fc1Tensor, dfc1relu));

		// FC1 layer
		// Compute derivative with respect to weights: gfc1 = (pool2 * dfc1relu')
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_fc1.inputs, ref_fc1.outputs, m_batchSize,
			&alpha, pool2, ref_fc1.inputs, dfc1relu, ref_fc1.outputs, &beta, gfc1, ref_fc1.inputs));
		// Compute derivative with respect to bias: gfc1bias = dfc1relu * 1_vec
		checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_fc1.outputs, m_batchSize,
			&alpha, dfc1relu, ref_fc1.outputs, onevec, 1, &beta, gfc1bias, 1));
		// Compute derivative with respect to data (for previous layer): pfc1*dfc1relu (800x500*500xN)
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_fc1.inputs, m_batchSize, ref_fc1.outputs,
			&alpha, pfc1, ref_fc1.inputs, dfc1relu, ref_fc1.outputs, &beta, dfc1, ref_fc1.inputs));

		// Pool2 layer
		checkCUDNN(hipdnnPoolingBackward(cudnnHandle, poolDesc, &alpha,
			pool2Tensor, pool2, pool2Tensor, dfc1,
			conv2Tensor, conv2, &beta, conv2Tensor, dpool2));

		// Conv2 layer
		checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, conv2Tensor,
			dpool2, &beta, conv2BiasTensor, gconv2bias));


		checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, pool1Tensor,
			pool1, conv2Tensor, dpool2, conv2Desc,
			&beta, conv2filterDesc, gconv2));

		checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle, &alpha, conv2filterDesc,
			pconv2, conv2Tensor, dpool2, conv2Desc,
			&beta, pool1Tensor, dconv2));

		// Pool1 layer
		checkCUDNN(hipdnnPoolingBackward(cudnnHandle, poolDesc, &alpha,
			pool1Tensor, pool1, pool1Tensor, dconv2,
			conv1Tensor, conv1, &beta, conv1Tensor, dpool1));

		// Conv1 layer
		checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, conv1Tensor,
			dpool1, &beta, conv1BiasTensor, gconv1bias));

		checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, dataTensor,
			data, conv1Tensor, dpool1, conv1Desc,
			&beta, conv1filterDesc, gconv1));

		// No need for convBackwardData because there are no more layers below
	}

	void UpdateWeights(float learning_rate,
		ConvBiasLayer& conv1, ConvBiasLayer& conv2,
		float* pconv1, float* pconv1bias,
		float* pconv2, float* pconv2bias,
		float* pfc1, float* pfc1bias,
		float* pfc2, float* pfc2bias,
		float* gconv1, float* gconv1bias,
		float* gconv2, float* gconv2bias,
		float* gfc1, float* gfc1bias,
		float* gfc2, float* gfc2bias) {
		float alpha = -learning_rate;

		checkCudaErrors(hipSetDevice(m_gpuid));

		// Conv1
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(conv1.pconv.size()),
			&alpha, gconv1, 1, pconv1, 1));
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(conv1.pbias.size()),
			&alpha, gconv1bias, 1, pconv1bias, 1));

		// Conv2
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(conv2.pconv.size()),
			&alpha, gconv2, 1, pconv2, 1));
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(conv2.pbias.size()),
			&alpha, gconv2bias, 1, pconv2bias, 1));

		// Fully connected 1
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc1.pneurons.size()),
			&alpha, gfc1, 1, pfc1, 1));
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc1.pbias.size()),
			&alpha, gfc1bias, 1, pfc1bias, 1));

		// Fully connected 2
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc2.pneurons.size()),
			&alpha, gfc2, 1, pfc2, 1));
		checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc2.pbias.size()),
			&alpha, gfc2bias, 1, pfc2bias, 1));
	}
};


class Verifier {
public:
	Verifier(string);
	~Verifier();
	void verify(unsigned char* data, double *& result);
private:
	int width = 28;
	int height = 28;
	int channels = 1;
	ConvBiasLayer conv1;
	MaxPoolLayer pool1;
	ConvBiasLayer conv2;
	MaxPoolLayer pool2;
	FullyConnectedLayer fc1, fc2;
	TrainingContext test_context;
	float *d_data, *d_conv1, *d_pool1, *d_conv2, *d_pool2, *d_fc1, *d_fc1relu, *d_fc2, *d_fc2smax;
	float *d_pconv1, *d_pconv1bias, *d_pconv2, *d_pconv2bias;
	float *d_pfc1, *d_pfc1bias, *d_pfc2, *d_pfc2bias;
	float* d_onevec;
	void * d_cudnn_workspace;
};

/**
	status inference(char * data, char * &result)
	@return	: ���ش����룬���û�д���ͷ���0
	@data	: ͼ�����飬����Ϊ28 * 28 = 784
	@result	: ����һ����ַ�������ַ�д����10��float����Ӧ�˷ֱ�����0~9�ĸ��ʣ���ߵ�һ����Ϊ�жϵĽ��
*/
JNIEXPORT jint JNICALL Java_cunist_CalculateByGPU_inference
(JNIEnv * env, jobject obj, jbyteArray data, jdoubleArray result) {
	void * temp = (env->GetByteArrayElements(data, NULL));
	unsigned char * cData = static_cast<unsigned char*>(temp);
	double* cResult = env->GetDoubleArrayElements(result, NULL);
	static Verifier verifier(DATA_FILE_NAME);
	verifier.verify(cData, cResult);
	env->ReleaseDoubleArrayElements(result, cResult, 0);
	return 0;
}

void loadFromFile(std::string filename,
	float* d_pconv1, float* d_pconv1bias,
	float* d_pconv2, float* d_pconv2bias,
	float* d_pfc1, float* d_pfc1bias,
	float* d_pfc2, float* d_pfc2bias,
	float* d_onevec) {
	FILE* dataFile;
	/*if ((dataFile = fopen((filename + ".dat").c_str(), "rb")) == nullptr) {
		
	}*/
	dataFile = fopen(DATA_FILE_NAME.c_str(), "rb");
	float* temp = new float[400000];

#define loadTempFromFile(count) fread(temp, sizeof(float), count, dataFile)
#define copyToHost(dst, count) hipMemcpy(dst, temp, count*sizeof(float), hipMemcpyHostToDevice);


	loadTempFromFile(500);
	copyToHost(d_pconv1, 500);
	loadTempFromFile(20);
	copyToHost(d_pconv1bias, 20);
	loadTempFromFile(25000);
	copyToHost(d_pconv2, 25000);
	loadTempFromFile(50);
	copyToHost(d_pconv2bias, 50);
	loadTempFromFile(400000);
	copyToHost(d_pfc1, 400000);
	loadTempFromFile(500);
	copyToHost(d_pfc1bias, 500);
	loadTempFromFile(5000);
	copyToHost(d_pfc2, 5000);
	loadTempFromFile(10);
	copyToHost(d_pfc2bias, 10);
	loadTempFromFile(64);
	copyToHost(d_onevec, 64);


	fclose(dataFile);
	delete[] temp;
}

Verifier::Verifier(string fileName) : conv1((int)channels, 20, 5, (int)width, (int)height), 
	pool1(2, 2),
	conv2(conv1.out_channels, 50, 5, conv1.out_width / pool1.stride, conv1.out_height / pool1.stride),
	pool2(2, 2),
	fc1((conv2.out_channels * conv2.out_width * conv2.out_height) / (pool2.stride * pool2.stride),
	    500),
	fc2(fc1.outputs, 10),
	test_context(FLAGS_gpu, 1, conv1, pool1, conv2, pool2, fc1, fc2)
{
	checkCudaErrors(hipMalloc(&d_data, 200704));
	checkCudaErrors(hipMalloc(&d_conv1, 2949120));
	checkCudaErrors(hipMalloc(&d_pool1, 737280));
	checkCudaErrors(hipMalloc(&d_conv2, 819200));
	checkCudaErrors(hipMalloc(&d_pool2, 204800));
	checkCudaErrors(hipMalloc(&d_fc1, 128000));
	checkCudaErrors(hipMalloc(&d_fc1relu, 128000));
	checkCudaErrors(hipMalloc(&d_fc2, 128000));
	checkCudaErrors(hipMalloc(&d_fc2smax, 128000));

	checkCudaErrors(hipMalloc(&d_pconv1, sizeof(float) * 500));
	checkCudaErrors(hipMalloc(&d_pconv1bias, sizeof(float) * 20));
	checkCudaErrors(hipMalloc(&d_pconv2, sizeof(float) * 25000));
	checkCudaErrors(hipMalloc(&d_pconv2bias, sizeof(float) * 50));
	checkCudaErrors(hipMalloc(&d_pfc1, sizeof(float) * 400000));
	checkCudaErrors(hipMalloc(&d_pfc1bias, sizeof(float) * 500));
	checkCudaErrors(hipMalloc(&d_pfc2, sizeof(float) * 5000));
	checkCudaErrors(hipMalloc(&d_pfc2bias, sizeof(float) * 10));
	checkCudaErrors(hipMalloc(&d_onevec, sizeof(float) * 64));
	checkCudaErrors(hipMalloc(&d_cudnn_workspace, 3464));
	loadFromFile(fileName, d_pconv1, d_pconv1bias, d_pconv2, d_pconv2bias, d_pfc1, d_pfc1bias, d_pfc2, d_pfc2bias, d_onevec);
}

Verifier::~Verifier() {
	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(d_conv1));
	checkCudaErrors(hipFree(d_pool1));
	checkCudaErrors(hipFree(d_conv2));
	checkCudaErrors(hipFree(d_pool2));
	checkCudaErrors(hipFree(d_fc1));
	checkCudaErrors(hipFree(d_fc2));

	checkCudaErrors(hipFree(d_pconv1));
	checkCudaErrors(hipFree(d_pconv1bias));
	checkCudaErrors(hipFree(d_pconv2));
	checkCudaErrors(hipFree(d_pconv2bias));
	checkCudaErrors(hipFree(d_pfc1));
	checkCudaErrors(hipFree(d_pfc1bias));
	checkCudaErrors(hipFree(d_pfc2));
	checkCudaErrors(hipFree(d_pfc2bias));
	checkCudaErrors(hipFree(d_onevec));
}

void Verifier::verify(unsigned char* data, double*& result) {
	std::vector<float> temp(width * height);
	for (int i = 0; i < height*width; ++i)
		temp[i] = (float)data[i] / 255.0f;
	checkCudaErrors(hipMemcpyAsync(d_data, &temp[0], sizeof(float) * width * height, hipMemcpyHostToDevice));
	test_context.ForwardPropagation(d_data, d_conv1, d_pool1, d_conv2, d_pool2, d_fc1, d_fc1relu, d_fc2, d_fc2smax,
		d_pconv1, d_pconv1bias, d_pconv2, d_pconv2bias, d_pfc1, d_pfc1bias,
		d_pfc2, d_pfc2bias, d_cudnn_workspace, d_onevec);
	std::vector<float> class_vec(10);
	checkCudaErrors(hipMemcpy(&class_vec[0], d_fc2smax, sizeof(float) * 10, hipMemcpyDeviceToHost));
	for (int i = 0; i < 10; ++i)
		result[i] = (double)class_vec[i];
}
